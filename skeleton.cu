#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>

#define THREADS_PER_BLOCK 32
#define BLOCKS_PER_SM 1

#define KB(x) ((x) << 10)
#define MB(x) ((x) << 20)

#define CACHE_BLOCK_SIZE 64

__global__ void test_kernel(char *data, int maxIdx)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	while(tid < maxIdx) {
		data[tid] = tid;
		tid += blockDim.x * gridDim.x;
	}
}

int main()
{
	char *data;

	for (int i = 1; i < 256; i++) {
		int maxIdx = MB(i);
		float time_elapsed, total_time = 0;
		hipEvent_t before, after;
		hipEventCreate(&before);
		hipEventCreate(&after);
		hipMallocManaged(&data, MB(i));

#if 1
		for (int j = 0; j < MB(i); j+=CACHE_BLOCK_SIZE)
			data[j] = j;
#endif
		for (int tries = 0; tries < 5; tries++) {
			hipEventRecord(before, 0);
			test_kernel<<<BLOCKS_PER_SM, THREADS_PER_BLOCK>>>(data, maxIdx);
			hipDeviceSynchronize();
			hipEventRecord(after, 0);

			hipEventSynchronize(before);
			hipEventSynchronize(after);
			hipEventElapsedTime(&time_elapsed, before, after);
			total_time += time_elapsed;
		}
		std::cout << i << "," << total_time/5 << "," << total_time/(5*i) << std::endl;
		hipFree(data);
	}

	return 0;
}
