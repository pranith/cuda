#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>

#define THREADS_PER_BLOCK 32
#define BLOCKS_PER_SM 6

#define KB(x) ((x) << 10)
#define MB(x) ((x) << 20)

__global__ void test_kernel(char *data, int maxIdx)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	while(tid < maxIdx) {
		data[tid] = tid;
		tid += blockDim.x * gridDim.x;
	}
}

int main()
{
	char *data;

	for (int i = 1; i < 256; i++) {
		int maxIdx = MB(i);
		float time_elapsed, total_time = 0;
		hipEvent_t before, after;
		hipEventCreate(&before);
		hipEventCreate(&after);
		hipMallocManaged(&data, MB(i));
		for (int tries = 0; tries < 5; tries++) {
			hipEventRecord(before, 0);
			test_kernel<<<BLOCKS_PER_SM, THREADS_PER_BLOCK>>>(data, maxIdx);
			hipDeviceSynchronize();
			hipEventRecord(after, 0);

			hipEventSynchronize(before);
			hipEventSynchronize(after);
			hipEventElapsedTime(&time_elapsed, before, after);
			total_time += time_elapsed;
		}
		std::cout << i << "," << total_time/5 << std::endl;
		hipFree(data);
	}

	return 0;
}

